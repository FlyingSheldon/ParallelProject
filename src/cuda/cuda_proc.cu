#include "hip/hip_runtime.h"
#include "cuda_proc.h"
#include "proc.cuh"
#include <cstdio>
#include <vector>

void cudaSayHi() { sayHi<<<1, 2>>>(); }

void cudaBrighten(uint8_t *img, size_t size, size_t pixelSize, double value) {
  uint8_t *deviceImg;
  hipMalloc(&deviceImg, size * pixelSize * sizeof(uint8_t));

  hipMemcpy(deviceImg, img, size * pixelSize * sizeof(uint8_t),
             hipMemcpyHostToDevice);

  size_t blocksPerGrid =
      (size * pixelSize + kThreadPerBlock - 1) / kThreadPerBlock;
  brighten<<<blocksPerGrid, kThreadPerBlock>>>(deviceImg, size, pixelSize,
                                               value);
  hipMemcpy(img, deviceImg, size * pixelSize * sizeof(uint8_t),
             hipMemcpyDeviceToHost);
  hipFree(deviceImg);
}

void cudaEdgeLPFDbg(uint8_t *edges, uint8_t *output, size_t width,
                    size_t height, int lpf) {
  size_t size = width * height;
  uint8_t *deviceEdges, *deviceLPF;

  hipMalloc(&deviceEdges, size * sizeof(uint8_t));
  hipMalloc(&deviceLPF, size * sizeof(uint8_t));

  hipMemcpy(deviceEdges, edges, size * sizeof(uint8_t),
             hipMemcpyHostToDevice);

  dim3 blockDim(kBlockEdgeSize, kBlockEdgeSize, 1);
  dim3 gridDim((width + (blockDim.x - 2) - 1) / (blockDim.x - 2),
               (height + (blockDim.y - 2) - 1) / (blockDim.y - 2));

  edgeLPFDbg<<<gridDim, blockDim>>>(deviceEdges, deviceLPF, width, height, lpf);

  hipMemcpy(output, deviceLPF, size, hipMemcpyDeviceToHost);
  hipMemcpy(edges, deviceEdges, size, hipMemcpyDeviceToHost);
}

void cudaEdgeDetect(uint8_t *img, uint8_t *edges, size_t pixelSize,
                    size_t width, size_t height, double eth, double *hsv) {
  size_t size = width * height;
  uint8_t *deviceImg, *deviceEdges;
  size_t blockPerGrid = (size + kThreadPerBlock - 1) / kThreadPerBlock;

  double *deviceHsv;
  ValueMinMaxSum *deviceVmms;
  ValueMinMaxSum initialVmms;
  initialVmms.max = 0;
  initialVmms.min = 255;
  initialVmms.sum = 0;

  hipMalloc(&deviceImg, size * pixelSize * sizeof(uint8_t));
  hipMalloc(&deviceHsv, size * 3 * sizeof(double));
  hipMalloc(&deviceVmms, sizeof(ValueMinMaxSum));
  hipMalloc(&deviceEdges, size * sizeof(uint8_t));

  hipMemcpy(deviceImg, img, size * pixelSize * sizeof(uint8_t),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceVmms, &initialVmms, sizeof(ValueMinMaxSum),
             hipMemcpyHostToDevice);

  dim3 blockDim(kBlockEdgeSize, kBlockEdgeSize, 1);
  dim3 gridDim((width + (blockDim.x - 2) - 1) / (blockDim.x - 2),
               (height + (blockDim.y - 2) - 1) / (blockDim.y - 2));

  rgbToHsvAndDeltaReduce<kThreadPerBlock><<<blockPerGrid, kThreadPerBlock>>>(
      deviceImg, deviceHsv, size, deviceVmms);

  hipDeviceSynchronize();

  edgeDetect<<<gridDim, blockDim>>>(deviceHsv, deviceEdges, width, height, eth);

  hipMemcpy(edges, deviceEdges, size * sizeof(uint8_t),
             hipMemcpyDeviceToHost);

  if (hsv) {
    hipMemcpy(hsv, deviceHsv, size * pixelSize * sizeof(double),
               hipMemcpyDeviceToHost);
  }
}

void cudaSharpen(uint8_t *img, size_t pixelSize, size_t width, size_t height,
                 double value, double eth, int lpf, double *hsv) {
  size_t size = width * height;
  uint8_t *deviceImg, *deviceEdges;
  size_t blockPerGrid = (size + kThreadPerBlock - 1) / kThreadPerBlock;

  double *deviceHsv;

  ValueMinMaxSum *deviceVmms;
  ValueMinMaxSum initialVmms;
  initialVmms.max = 0;
  initialVmms.min = 255;
  initialVmms.sum = 0;

  hipMalloc(&deviceImg, size * pixelSize * sizeof(uint8_t));
  hipMalloc(&deviceHsv, size * 3 * sizeof(double));
  hipMalloc(&deviceVmms, sizeof(ValueMinMaxSum));
  hipMalloc(&deviceEdges, size * sizeof(uint8_t));

  hipMemcpy(deviceImg, img, size * pixelSize * sizeof(uint8_t),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceVmms, &initialVmms, sizeof(ValueMinMaxSum),
             hipMemcpyHostToDevice);

  dim3 blockDim(kBlockEdgeSize, kBlockEdgeSize, 1);
  dim3 gridDim((width + (blockDim.x - 2) - 1) / (blockDim.x - 2),
               (height + (blockDim.y - 2) - 1) / (blockDim.y - 2));

  rgbToHsvAndDeltaReduce<kThreadPerBlock><<<blockPerGrid, kThreadPerBlock>>>(
      deviceImg, deviceHsv, size, deviceVmms);

  hipDeviceSynchronize();

  edgeDetect<<<gridDim, blockDim>>>(deviceHsv, deviceEdges, width, height, eth);

  hipDeviceSynchronize();

  edgeSharpen<<<gridDim, blockDim>>>(deviceHsv, width, height, value, eth, lpf,
                                     deviceEdges, deviceVmms, deviceImg);

  hipMemcpy(img, deviceImg, size * pixelSize * sizeof(uint8_t),
             hipMemcpyDeviceToHost);
  hipMemcpy(hsv, deviceHsv, size * pixelSize * sizeof(double),
             hipMemcpyDeviceToHost);

  hipFree(deviceImg);
  hipFree(deviceEdges);
  hipFree(deviceHsv);
  hipFree(deviceVmms);
}
